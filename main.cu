#include <iostream>
#include "pyramNeuron.cu"
#include "io.cpp"

int main(void)
{
    const int N = 1<<10;
    const int BLOCK = 256;
    const int ITERS = 1000;

    pyramNeuron *vpn, *d_vpn;
    vpn = (pyramNeuron*)malloc(N*sizeof(pyramNeuron));
    hipMalloc(&d_vpn, N*sizeof(pyramNeuron));

    pyramReceptor *vpr, *d_vpr;
    vpr = (pyramReceptor*)malloc(N*sizeof(pyramReceptor));
    hipMalloc(&d_vpr, N*sizeof(pyramReceptor));

    for (int i = 0; i < N; i++) initPyramNeuron(&vpn[i]);
    for (int i = 0; i < N; i++) initPyramReceptor(&vpr[i]);

    pyramNeuron* result;
    result = (pyramNeuron*)malloc(ITERS*sizeof(pyramNeuron));
    for(int i=0; i<ITERS; i++){
      hipMemcpy(d_vpr, vpr, N*sizeof(pyramReceptor), hipMemcpyHostToDevice);
      hipMemcpy(d_vpn, vpn, N*sizeof(pyramNeuron), hipMemcpyHostToDevice);
      
      pyramStep<<<(N+BLOCK-1)/BLOCK, BLOCK>>>(N, 0.1f, d_vpn, d_vpr);

      hipMemcpy(vpn, d_vpn, N*sizeof(pyramNeuron), hipMemcpyDeviceToHost);
      hipMemcpy(vpr, d_vpr, N*sizeof(pyramReceptor), hipMemcpyDeviceToHost);
      
      memcpy(&result[i],&vpn[0],sizeof(pyramNeuron));
    }

    pyramToJson(ITERS,result,"result.json");

    hipFree(d_vpn);
    free(vpn);
    hipFree(d_vpr);
    free(vpr);
    free(result);
}