#include "headers.h"

int main(void)
{
    const int N = 1<<10;
    const int BLOCK = 256;
    const int ITERS = 10000;

    interNeuron *vin, *d_vin;
    vin = (interNeuron*)malloc(N*sizeof(interNeuron));
    hipMalloc(&d_vin, N*sizeof(interNeuron));

    interReceptor *vir, *d_vir;
    vir = (interReceptor*)malloc(N*sizeof(interReceptor));
    hipMalloc(&d_vir, N*sizeof(interReceptor));

    for (int i = 0; i < N; i++) inter::initInterNeuron(&vin[i]);
    for (int i = 0; i < N; i++) inter::initInterReceptor(&vir[i]);

    interNeuron* result;
    result = (interNeuron*)malloc(ITERS*sizeof(interNeuron));
    for(int i=0; i<ITERS; i++){
      if(i==200){
        for(int i=0; i<N; i++) vir[i].g_E+=1;
      }

      hipMemcpy(d_vir, vir, N*sizeof(interReceptor), hipMemcpyHostToDevice);
      hipMemcpy(d_vin, vin, N*sizeof(interNeuron), hipMemcpyHostToDevice);

      inter::interStep<<<(N+BLOCK-1)/BLOCK, BLOCK>>>(N, 0.1f, d_vin, d_vir);

      hipMemcpy(vin, d_vin, N*sizeof(interNeuron), hipMemcpyDeviceToHost);
      hipMemcpy(vir, d_vir, N*sizeof(interReceptor), hipMemcpyDeviceToHost);

      memcpy(&result[i],&vin[0],sizeof(interNeuron));
    }

    interToJson(ITERS,result,"result.json");

    hipFree(d_vin);
    free(vin);
    hipFree(d_vir);
    free(vir);
    free(result);
}