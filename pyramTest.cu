#include <iostream>
#include "pyramNeuron.cu"
#include "interNeuron.cu"
#include "io.cpp"

int main(void)
{
    const int N = 1<<10;
    const int BLOCK = 256;
    const int ITERS = 40000;

    pyramNeuron *vpn, *d_vpn;
    vpn = (pyramNeuron*)malloc(N*sizeof(pyramNeuron));
    hipMalloc(&d_vpn, N*sizeof(pyramNeuron));

    pyramReceptor *vpr, *d_vpr;
    vpr = (pyramReceptor*)malloc(N*sizeof(pyramReceptor));
    hipMalloc(&d_vpr, N*sizeof(pyramReceptor));

    for (int i = 0; i < N; i++) pyram::initPyramNeuron(&vpn[i]);
    for (int i = 0; i < N; i++) pyram::initPyramReceptor(&vpr[i]);

    pyramNeuron* result;
    result = (pyramNeuron*)malloc(ITERS*sizeof(pyramNeuron));
    for(int i=0; i<ITERS; i++){
      // if(i==200){
      //   for(int i=0; i<N; i++) vpr[i].g_E+=0.275;
      // }

      hipMemcpy(d_vpr, vpr, N*sizeof(pyramReceptor), hipMemcpyHostToDevice);
      hipMemcpy(d_vpn, vpn, N*sizeof(pyramNeuron), hipMemcpyHostToDevice);
      
      pyram::pyramStep<<<(N+BLOCK-1)/BLOCK, BLOCK>>>(N, i, 0.1f, d_vpn, d_vpr);

      hipMemcpy(vpn, d_vpn, N*sizeof(pyramNeuron), hipMemcpyDeviceToHost);
      hipMemcpy(vpr, d_vpr, N*sizeof(pyramReceptor), hipMemcpyDeviceToHost);

      memcpy(&result[i],&vpn[0],sizeof(pyramNeuron));
    }

    pyramToJson(ITERS,result,"result.json");

    hipFree(d_vpn);
    free(vpn);
    hipFree(d_vpr);
    free(vpr);
    free(result);
}